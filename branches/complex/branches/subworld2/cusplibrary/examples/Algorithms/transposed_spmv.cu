
/*****************************************************************************
*
* Copyright (c) 2014-2015 by University of Queensland
* http://www.uq.edu.au
*
* Primary Business: Queensland, Australia
* Licensed under the Open Software License version 3.0
* http://www.opensource.org/licenses/osl-3.0.php
*
* Development until 2012 by Earth Systems Science Computational Center (ESSCC)
* Development 2012-2013 by School of Earth Sciences
* Development from 2014 by Centre for Geoscience Computing (GeoComp)
*
*****************************************************************************/

#include <cusp/multiply.h>
#include <cusp/cds_matrix.h>
#include <cusp/print.h>

typedef cusp::device_memory MemorySpace;
typedef double ValueType;

int main(void)
{
    // initialize matrix
    // args are: num_rows, num_entries, num_diagonals, blocksize
    cusp::cds_matrix<int, ValueType, MemorySpace> A(8,28,3,2);
    A.diagonal_offsets[0] = -3;
    A.diagonal_offsets[1] = 0;
    A.diagonal_offsets[2] = 2;

    A.values(6,0) = 11;
    A.values(7,0) = 17;
    A.values(6,1) = 23;
    A.values(7,1) = 27;

    A.values(0,2) =  1;
    A.values(1,2) =  3;
    A.values(0,3) =  5;
    A.values(1,3) =  7;
    A.values(2,2) =  8;
    A.values(3,2) = 29;
    A.values(2,3) = 31;
    A.values(3,3) = 37;
    A.values(4,2) = 41;
    A.values(5,2) = -1;
    A.values(4,3) = -3;
    A.values(5,3) = -5;
    A.values(6,2) = -7;
    A.values(7,2) =-11;
    A.values(6,3) =-13;
    A.values(7,3) =-17;

    A.values(0,4) = 21;
    A.values(1,4) = 12;
    A.values(0,5) = 14;
    A.values(1,5) = -3;
    A.values(2,4) = -7;
    A.values(3,4) = 22;
    A.values(2,5) =-31;
    A.values(3,5) =  5;

    // initialize input vector
    cusp::array1d<ValueType, MemorySpace> x(8);
    x[0] = 1;
    x[1] = 2;
    x[2] = 3;
    x[3] = 5;
    x[4] =-1;
    x[5] =-3;
    x[6] =-7;
    x[7] =-5;

    // allocate output vector
    cusp::array1d<ValueType, MemorySpace> y(8);

    // compute y = A * x
    cusp::multiply(A, x, y);

    // print y
    cusp::print(y);

    cusp::transposed_multiply(A, y, x);

    // print x
    cusp::print(x);

    return 0;
}

