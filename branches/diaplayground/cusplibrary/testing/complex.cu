#include "hip/hip_runtime.h"
#include <unittest/unittest.h>
#include <cusp/array1d.h>
#include <cusp/complex.h>
#include <complex>

#include <cusp/verify.h>

#define ASSERT_COMPLEX_ALMOST_EQUAL(X,Y) {unittest::assert_almost_equal((X.real()),(Y.real()), __FILE__, __LINE__);unittest::assert_almost_equal((X.imag()),(Y.imag()), __FILE__, __LINE__);}

template< typename T1, typename T2 >
struct is_same_type      { enum { result = false }; };

template< typename T>
struct is_same_type<T,T> { enum { result = true }; };

#ifdef __GNUC__
extern "C"{
  float __complex__ cacosf(float __complex__ z);
  double __complex__ cacos(double __complex__ z);
  float __complex__ casinf(float __complex__ z);
  double __complex__ casin(double __complex__ z);
  float __complex__ catanf(float __complex__ z);
  double __complex__ catan(double __complex__ z);
  float __complex__ cacoshf(float __complex__ z);
  double __complex__ cacosh(double __complex__ z);
  float __complex__ casinhf(float __complex__ z);
  double __complex__ casinh(double __complex__ z);
  float __complex__ catanhf(float __complex__ z);
  double __complex__ catanh(double __complex__ z);
  double creal(double __complex__ z);
  double cimag(double __complex__ z);
}
#endif


// Macro to create host and device versions of a unit test
#define DECLARE_NUMERIC_UNITTEST(VTEST)                    \
void VTEST##Float(void)   {  VTEST<float>(); }  \
void VTEST##Double(void)   {  VTEST<double>(); }  \
DECLARE_UNITTEST(VTEST##Float);                                 \
DECLARE_UNITTEST(VTEST##Double);

template <typename ValueType>
__host__ bool compareWithStd(cusp::complex<ValueType> a){
  //  std::cout << "Testing " << a << std::endl; 
  cusp::complex<ValueType> b(a.real(),a.imag());
  std::complex<ValueType> s_a(a.real(),a.imag());
  std::complex<ValueType> s_b(b.real(),b.imag());
  ASSERT_COMPLEX_ALMOST_EQUAL(a,s_a);
  ASSERT_COMPLEX_ALMOST_EQUAL(b,s_b);
  b += a;
  s_b += s_a;
  ASSERT_COMPLEX_ALMOST_EQUAL(b,s_b);
  b -= a;
  s_b -= s_a;
  ASSERT_COMPLEX_ALMOST_EQUAL(b,s_b);
  b *= a;
  s_b *= s_a;
  ASSERT_COMPLEX_ALMOST_EQUAL(b,s_b);
  b /= a;
  s_b /= s_a;
  ASSERT_COMPLEX_ALMOST_EQUAL(b,s_b);

  b = a*ValueType(2.0);
  s_b = s_a*ValueType(2.0);
  ASSERT_COMPLEX_ALMOST_EQUAL(b,s_b);
  b = ValueType(2.0)*a;
  s_b = ValueType(2.0)*s_a;
  ASSERT_COMPLEX_ALMOST_EQUAL(b,s_b);
  b = b*a;
  s_b = s_b*s_a;
  ASSERT_COMPLEX_ALMOST_EQUAL(b,s_b);
  b = a/ValueType(2.0);
  s_b = s_a/ValueType(2.0);
  ASSERT_COMPLEX_ALMOST_EQUAL(b,s_b);
  b = ValueType(2.0)/a;
  s_b = ValueType(2.0)/s_a;
  ASSERT_COMPLEX_ALMOST_EQUAL(b,s_b);
  b = a/b;
  s_b = s_a/s_b;
  ASSERT_COMPLEX_ALMOST_EQUAL(b,s_b);
  b = a-ValueType(2.0);
  s_b = s_a-ValueType(2.0);
  ASSERT_COMPLEX_ALMOST_EQUAL(b,s_b);
  b = ValueType(2.0)-a;
  s_b = ValueType(2.0)-s_a;
  ASSERT_COMPLEX_ALMOST_EQUAL(b,s_b);
  b = a-b;
  s_b = s_a-s_b;
  ASSERT_COMPLEX_ALMOST_EQUAL(b,s_b);
  b = a+ValueType(2.0);
  s_b = s_a+ValueType(2.0);
  ASSERT_COMPLEX_ALMOST_EQUAL(b,s_b);
  b = ValueType(2.0)+a;
  s_b = ValueType(2.0)+s_a;
  ASSERT_COMPLEX_ALMOST_EQUAL(b,s_b);
  b = a+b;
  s_b = s_a+s_b;
  ASSERT_COMPLEX_ALMOST_EQUAL(b,s_b);

  b = cusp::abs(a);
  s_b = std::abs(s_a);
  ASSERT_COMPLEX_ALMOST_EQUAL(b,s_b);
  b = cusp::arg(a);
  s_b = std::arg(s_a);
  ASSERT_COMPLEX_ALMOST_EQUAL(b,s_b);
  b = cusp::norm(a);
  s_b = std::norm(s_a);
  ASSERT_COMPLEX_ALMOST_EQUAL(b,s_b);
  b = cusp::conj(a);
  s_b = std::conj(s_a);
  ASSERT_COMPLEX_ALMOST_EQUAL(b,s_b);
  b = cusp::polar(norm(a),a.imag());
  s_b = std::polar(norm(a),a.imag());
  ASSERT_COMPLEX_ALMOST_EQUAL(b,s_b);

  b = cusp::cos(a);
  s_b = std::cos(s_a);
  ASSERT_COMPLEX_ALMOST_EQUAL(b,s_b);
  b = cusp::cosh(a);
  s_b = std::cosh(s_a);
  ASSERT_COMPLEX_ALMOST_EQUAL(b,s_b);
  b = cusp::exp(a);
  s_b = std::exp(s_a);
  ASSERT_COMPLEX_ALMOST_EQUAL(b,s_b);

  b = cusp::log(a);
  s_b = std::log(s_a);
  ASSERT_COMPLEX_ALMOST_EQUAL(b,s_b);
  b = cusp::log10(a);
  s_b = std::log10(s_a);
  ASSERT_COMPLEX_ALMOST_EQUAL(b,s_b);

  b = cusp::pow(a,b);
  s_b = std::pow(s_a,s_b);
  ASSERT_COMPLEX_ALMOST_EQUAL(b,s_b);
  b = cusp::pow(a,ValueType(1.3));
  s_b = std::pow(s_a,ValueType(1.3));
  ASSERT_COMPLEX_ALMOST_EQUAL(b,s_b);
  b = cusp::pow(ValueType(1.4),a);
  s_b = std::pow(ValueType(1.4),s_a);
  ASSERT_COMPLEX_ALMOST_EQUAL(b,s_b);
  // Missing int implentation in std
  b = cusp::pow(a,4);
  s_b = std::pow(s_a,ValueType(4.0));
  ASSERT_COMPLEX_ALMOST_EQUAL(b,s_b);
  b = cusp::sin(a);
  s_b = std::sin(s_a);
  ASSERT_COMPLEX_ALMOST_EQUAL(b,s_b);
  b = cusp::sinh(a);
  s_b = std::sinh(s_a);
  ASSERT_COMPLEX_ALMOST_EQUAL(b,s_b);
  b = cusp::sqrt(a);
  s_b = std::sqrt(s_a);
  ASSERT_COMPLEX_ALMOST_EQUAL(b,s_b);
  b = cusp::tan(a);  
  s_b = std::tan(s_a);
  ASSERT_COMPLEX_ALMOST_EQUAL(b,s_b);
  b = cusp::tanh(a);
  s_b = std::tanh(s_a);
  ASSERT_COMPLEX_ALMOST_EQUAL(b,s_b);

  // Inverse trigonometic functions not part of standard 
  /*
  a = cusp::acos(b);
  s_a = std::acos(s_b);
  ASSERT_COMPLEX_ALMOST_EQUAL(a,s_a);
  b = cusp::asin(a);
  s_b = std::asin(s_a);
  ASSERT_COMPLEX_ALMOST_EQUAL(b,s_b);
  a = cusp::atan(b);
  s_a = std::atan(s_b);
  ASSERT_COMPLEX_ALMOST_EQUAL(a,s_a);
  */

#ifdef __GNUC__  
  /* Use the c99 complex function from gcc to test the
   function not part of the standard */
  if(is_same_type<ValueType,float>::result){
    __complex__ float g_a;
    __complex__ float g_b;
    g_a = s_a.real() + s_a.imag()*__I__;
    g_b = s_b.real() + s_b.imag()*__I__;
    b = cusp::acos(a);
    g_b = cacosf(g_a);
    s_b = std::complex<ValueType>(creal(g_b),cimag(g_b));
    ASSERT_COMPLEX_ALMOST_EQUAL(b,s_b);

    g_a = s_a.real() + s_a.imag()*__I__;
    g_b = s_b.real() + s_b.imag()*__I__;
    b = cusp::asin(a);
    g_b = casinf(g_a);
    s_b = std::complex<ValueType>(creal(g_b),cimag(g_b));
    ASSERT_COMPLEX_ALMOST_EQUAL(b,s_b);

    g_a = s_a.real() + s_a.imag()*__I__;
    g_b = s_b.real() + s_b.imag()*__I__;
    b = cusp::atan(a);
    g_b = catanf(g_a);
    s_b = std::complex<ValueType>(creal(g_b),cimag(g_b));
    ASSERT_COMPLEX_ALMOST_EQUAL(b,s_b);

    g_a = s_a.real() + s_a.imag()*__I__;
    g_b = s_b.real() + s_b.imag()*__I__;
    b = cusp::acosh(a);
    g_b = cacoshf(g_a);
    s_b = std::complex<ValueType>(creal(g_b),cimag(g_b));
    ASSERT_COMPLEX_ALMOST_EQUAL(b,s_b);

    g_a = s_a.real() + s_a.imag()*__I__;
    g_b = s_b.real() + s_b.imag()*__I__;
    b = cusp::asinh(a);
    g_b = casinhf(g_a);
    s_b = std::complex<ValueType>(creal(g_b),cimag(g_b));
    ASSERT_COMPLEX_ALMOST_EQUAL(b,s_b);

    if(a != cusp::complex<ValueType>(1,0)){
      g_a = s_a.real() + s_a.imag()*__I__;
      g_b = s_b.real() + s_b.imag()*__I__;
      b = cusp::atanh(a);
      g_b = catanhf(g_a);
      s_b = std::complex<ValueType>(creal(g_b),cimag(g_b));
      ASSERT_COMPLEX_ALMOST_EQUAL(b,s_b);
    }
  }else if(is_same_type<ValueType,double>::result){
    __complex__ double g_a;
    __complex__ double g_b;
    g_a = s_a.real() + s_a.imag()*__I__;
    g_b = s_b.real() + s_b.imag()*__I__;
    b = cusp::acos(a);
    g_b = cacos(g_a);
    s_b = std::complex<ValueType>(creal(g_b),cimag(g_b));
    ASSERT_COMPLEX_ALMOST_EQUAL(b,s_b);

    g_a = s_a.real() + s_a.imag()*__I__;
    g_b = s_b.real() + s_b.imag()*__I__;
    b = cusp::asin(a);
    g_b = casin(g_a);
    s_b = std::complex<ValueType>(creal(g_b),cimag(g_b));
    ASSERT_COMPLEX_ALMOST_EQUAL(b,s_b);

    g_a = s_a.real() + s_a.imag()*__I__;
    g_b = s_b.real() + s_b.imag()*__I__;
    b = cusp::atan(a);
    g_b = catan(g_a);
    s_b = std::complex<ValueType>(creal(g_b),cimag(g_b));
    ASSERT_COMPLEX_ALMOST_EQUAL(b,s_b);

    g_a = s_a.real() + s_a.imag()*__I__;
    g_b = s_b.real() + s_b.imag()*__I__;
    b = cusp::acosh(a);
    g_b = cacosh(g_a);
    s_b = std::complex<ValueType>(creal(g_b),cimag(g_b));
    ASSERT_COMPLEX_ALMOST_EQUAL(b,s_b);

    g_a = s_a.real() + s_a.imag()*__I__;
    g_b = s_b.real() + s_b.imag()*__I__;
    b = cusp::asinh(a);
    g_b = casinh(g_a);
    s_b = std::complex<ValueType>(creal(g_b),cimag(g_b));
    ASSERT_COMPLEX_ALMOST_EQUAL(b,s_b);

    if(a != cusp::complex<ValueType>(1,0)){
      g_a = s_a.real() + s_a.imag()*__I__;
      g_b = s_b.real() + s_b.imag()*__I__;
      b = cusp::atanh(a);
      g_b = catanh(g_a);
      s_b = std::complex<ValueType>(creal(g_b),cimag(g_b));
      ASSERT_COMPLEX_ALMOST_EQUAL(b,s_b);
    }
  }
#endif
  return true;
}

template <typename ValueType>
__host__ __device__ cusp::complex<ValueType> test_complex_members(){
  cusp::complex<ValueType> a(ValueType(1.0),ValueType(1.0));
  cusp::complex<ValueType> b(ValueType(2.0),ValueType(1.0));
  a += b;
  a -= b;
  a *= b;
  a /= b;
  return a;
}


template <typename ValueType>
__host__ __device__ cusp::complex<ValueType> test_complex_non_members(){
  cusp::complex<ValueType> a(ValueType(3.0),ValueType(1.0));
  cusp::complex<ValueType> b(ValueType(2.0),ValueType(-1.0));
  a = a*ValueType(2.0);
  a = ValueType(2.0)*b;
  a = a*b;
  a = a/ValueType(2.0);
  a = ValueType(2.0)/b;
  a = a/b;
  a = a-ValueType(2.0);
  a = ValueType(2.0)-b;
  a = a-b;
  a = a+ValueType(2.0);
  a = ValueType(2.0)+b;
  a = a+b;
  b = cusp::abs(b);
  b = cusp::arg(b);
  b = cusp::norm(b);
  b = cusp::conj(b);
  b = cusp::polar(ValueType(0.3),ValueType(3.0));

  a = cusp::cos(b);
  b = cusp::cosh(a);
  a = cusp::exp(b);

  b = cusp::log(a);
  a = cusp::log10(b);

  b = cusp::pow(a,b);
  a = cusp::pow(b,ValueType(1.3));
  b = cusp::pow(ValueType(1.4),a);
  a = cusp::pow(b,4);
  b = cusp::sin(a);
  a = cusp::sinh(b);
  b = cusp::sqrt(a);
  a = cusp::tan(b);
  b = cusp::tanh(a);
  a = cusp::acos(b);
  b = cusp::asin(a);
  a = cusp::atan(b);
  return a;
}

template <typename ValueType>
__host__ __device__ cusp::complex<ValueType> test_complex_compilation_entry(){
  return test_complex_members<ValueType>() + test_complex_non_members<ValueType>();
}

template <typename ValueType>
__global__ void test_complex_compilation_kernel(cusp::complex<ValueType> * a){
  cusp::complex<ValueType> ret = test_complex_compilation_entry<ValueType>();
  *a = ret;
}

#if __CUDA_ARCH__ < 130
// Don't try to run the double precision tests if the compiled
// architecture doesn't support it 
template <>
__global__ void test_complex_compilation_kernel(cusp::complex<double> * a){
}
#endif

bool compiled_architecture_supports_double(void){
#if __CUDA_ARCH__ >= 130
  return true;
#else
  return false;
#endif
}

bool device_supports_double(void)
{
    int current_device = -1;
    hipDeviceProp_t properties;

    hipError_t error = hipGetDevice(&current_device);
    if(error)
        throw thrust::system_error(error, thrust::cuda_category());

    if(current_device < 0)
        throw thrust::system_error(hipErrorNoDevice, thrust::cuda_category());
    
    // the properties weren't found, ask the runtime to generate them
    error = hipGetDeviceProperties(&properties, current_device);

    if(error)
      throw thrust::system_error(error, thrust::cuda_category());

    return properties.major >= 2 || (properties.major == 1 && properties.minor >= 3);
}

template <typename MemorySpace>
void TestComplexRealConversion()
{
  typedef float                Real;
  typedef cusp::complex<float> Complex;

  cusp::array1d<Real, MemorySpace>    real_values(4);
  cusp::array1d<Complex, MemorySpace> complex_values(4);

  // test real->complex conversion
  real_values[0] = 0;
  real_values[1] = 1;
  real_values[2] = 2;
  real_values[3] = 3;
  
  complex_values = real_values;

  ASSERT_EQUAL((Complex) complex_values[0], Complex(0,0));
  ASSERT_EQUAL((Complex) complex_values[1], Complex(1,0));
  ASSERT_EQUAL((Complex) complex_values[2], Complex(2,0));
  ASSERT_EQUAL((Complex) complex_values[3], Complex(3,0));
}
DECLARE_HOST_DEVICE_UNITTEST(TestComplexRealConversion);


template <typename ValueType>
struct TestComplexStdComplexConversion
{
  void operator()(void)
  {
    typedef std::complex<ValueType>  StdComplex;
    typedef cusp::complex<ValueType> CuspComplex;

    ASSERT_EQUAL(CuspComplex(StdComplex(0,0)), CuspComplex(0,0));
    ASSERT_EQUAL(CuspComplex(StdComplex(0,1)), CuspComplex(0,1));
    ASSERT_EQUAL(CuspComplex(StdComplex(1,0)), CuspComplex(1,0));
    ASSERT_EQUAL(CuspComplex(StdComplex(1,2)), CuspComplex(1,2));

    // can't test StdComplex(CuspComplex(...)) due to constructor ambiguity

    { StdComplex a(0,0); CuspComplex b = a;  ASSERT_EQUAL(b, CuspComplex(0,0)); }
    { StdComplex a(0,1); CuspComplex b = a;  ASSERT_EQUAL(b, CuspComplex(0,1)); }
    { StdComplex a(1,0); CuspComplex b = a;  ASSERT_EQUAL(b, CuspComplex(1,0)); }
    { StdComplex a(1,2); CuspComplex b = a;  ASSERT_EQUAL(b, CuspComplex(1,2)); }

    { CuspComplex a(0,0); StdComplex b = a;  ASSERT_EQUAL(b, StdComplex(0,0)); }
    { CuspComplex a(0,1); StdComplex b = a;  ASSERT_EQUAL(b, StdComplex(0,1)); }
    { CuspComplex a(1,0); StdComplex b = a;  ASSERT_EQUAL(b, StdComplex(1,0)); }
    { CuspComplex a(1,2); StdComplex b = a;  ASSERT_EQUAL(b, StdComplex(1,2)); }
  }
};
SimpleUnitTest<TestComplexStdComplexConversion, unittest::type_list<float,double> > TestComplexStdComplexConversionInstance;


template <typename ValueType>
void TestComplex()
{
  cusp::complex<ValueType> a;
  cusp::complex<ValueType> * d_a;  
  hipMalloc(&d_a,sizeof(cusp::complex<ValueType>));
  test_complex_compilation_kernel<ValueType>
    <<<1,1>>>(d_a);
  hipMemcpy(&a,d_a,sizeof(cusp::complex<ValueType>),hipMemcpyDeviceToHost);
  std::complex<ValueType> b(a.real(),a.imag());
  a = test_complex_compilation_entry<ValueType>();
  // Don't check for equality between host and device code when the 
  // hardware device does not support double precision 
  if(is_same_type<ValueType,double>::result == false ||
     (device_supports_double() && compiled_architecture_supports_double())){
    ASSERT_COMPLEX_ALMOST_EQUAL(a,b);
  }
  // Test twice the unit circle 
  for(int i = 0;i < 32;i++){
    ValueType theta(ValueType(i*M_PI/8));
    compareWithStd<ValueType>(cusp::polar<ValueType>(ValueType(1),theta));
  }
}
DECLARE_NUMERIC_UNITTEST(TestComplex);


