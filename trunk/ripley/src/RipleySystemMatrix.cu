#include "hip/hip_runtime.h"

/*****************************************************************************
*
* Copyright (c) 2014 by University of Queensland
* http://www.uq.edu.au
*
* Primary Business: Queensland, Australia
* Licensed under the Open Software License version 3.0
* http://www.opensource.org/licenses/osl-3.0.php
*
* Development until 2012 by Earth Systems Science Computational Center (ESSCC)
* Development 2012-2013 by School of Earth Sciences
* Development from 2014 by Centre for Geoscience Computing (GeoComp)
*
*****************************************************************************/

#include "RipleySystemMatrix.h" 
#include "RipleyException.h" 

#include <esysUtils/index.h>
#include <escript/Data.h>
#include <escript/SolverOptions.h>

#include <cusp/multiply.h>
#include <cusp/krylov/bicgstab.h>
#include <cusp/krylov/cg.h>
#include <cusp/krylov/cgls.h>
#include <cusp/krylov/gmres.h>
#include <cusp/krylov/lsqr.h>
#include <cusp/precond/diagonal.h>

namespace bp = boost::python;

namespace ripley {

double gettime()
{
#ifdef _OPENMP
    return omp_get_wtime();
#else
    struct timeval tv;
    gettimeofday(&tv, NULL);
    suseconds_t ret = tv.tv_usec + tv.tv_sec*1e6;
    return 1e-6*(double)ret;
#endif
}

std::vector<int> SystemMatrix::cudaDevices;

void SystemMatrix::checkCUDA()
{
#ifdef USE_CUDA
    cudaDevices.clear();
    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);
    if (err == hipErrorInsufficientDriver) {
        std::cout << "Note: CUDA reported a driver error enumerating CUDA devices" << std::endl;
        cudaDevices.push_back(-1);
        return;
    } else if (deviceCount == 0 || err == hipErrorNoDevice) {
        std::cout << "Note: There is no device supporting CUDA" << std::endl;
        cudaDevices.push_back(-1);
        return;
    } else if (deviceCount == 1) {
        std::cout << "There is 1 GPU device" << std::endl;
    } else {
        std::cout << "There are " << deviceCount << " GPU devices" << std::endl;
    }

    for (int dev = 0; dev < deviceCount; ++dev) {
        hipDeviceProp_t deviceProp;
        CUDA_SAFE_CALL(hipGetDeviceProperties(&deviceProp, dev));

        std::cout << "\nDevice " << dev << ": \"" << deviceProp.name << "\" -- ";
        if (deviceProp.major == 9999 && deviceProp.minor == 9999) {
            std::cout << "No CUDA support." << std::endl;
        } else {
            cudaDevices.push_back(dev);
            std::cout << "Rev. " << deviceProp.major << "." << deviceProp.minor
                      << ", Mem: "
                      << deviceProp.totalGlobalMem << " bytes." << std::endl;
        }
    }
    // neither of the found devices supports CUDA:
    if (cudaDevices.empty()) {
        cudaDevices.push_back(-1);
    }
#endif
}

SystemMatrix::SystemMatrix(esysUtils::JMPI mpiInfo, int blocksize,
                           const escript::FunctionSpace& fs, int nRows,
                           const IndexVector& diagonalOffsets) :
    AbstractSystemMatrix(blocksize, fs, blocksize, fs),
    m_mpiInfo(mpiInfo)
{
    // count nonzero entries
    int numEntries = 0;
    for (size_t i = 0; i < diagonalOffsets.size(); i++) {
        numEntries += blocksize*blocksize*(nRows-std::abs(diagonalOffsets[i]));
    }

    mat.resize(nRows*blocksize, numEntries, diagonalOffsets.size(), blocksize);
    mat.diagonal_offsets.assign(diagonalOffsets.begin(), diagonalOffsets.end());
    matrixAltered = true;
}

void SystemMatrix::add(const IndexVector& rowIdx,
                       const std::vector<double>& array)
{
    const int blockSize = getBlockSize();
    const int emSize = rowIdx.size();
    for (int i=0; i<emSize; i++) {
        for (int j=0; j<emSize; j++) {
            const int revi = emSize-1-i;
            const int diag = j%2 + revi%2 + 3*(j/2+revi/2 + j/4+revi/4);
            for (int k=0; k<blockSize; k++) {
                for (int m=0; m<blockSize; m++) {
                    const int row = rowIdx[i]*blockSize + k;
                    const int d = diag*blockSize + m;
                    const int srcIdx = INDEX4(k, m, i, j, blockSize, blockSize, emSize);
                    mat.values(row, d) += array[srcIdx];
                }
            }
        }
    }
    matrixAltered = true;
}

void SystemMatrix::ypAx(escript::Data& y, escript::Data& x) const
{
    if (x.getDataPointSize() != getBlockSize()) {
        throw RipleyException("matrix vector product: block size does not match the number of components in input.");
    } else if (y.getDataPointSize() != getBlockSize()) {
        throw RipleyException("matrix vector product: block size does not match the number of components in output.");
    } else if (x.getFunctionSpace() != getColumnFunctionSpace()) {
        throw RipleyException("matrix vector product: matrix function space and function space of input don't match.");
    } else if (y.getFunctionSpace() != getRowFunctionSpace()) {
        throw RipleyException("matrix vector product: matrix function space and function space of output don't match.");
    }

    // expand data object if necessary to be able to grab the whole data
    const_cast<escript::Data*>(&x)->expand();
    y.expand();
    y.requireWrite();
    const double* x_dp = x.getSampleDataRO(0);
    double* y_dp = y.getSampleDataRW(0);
    //double T0 = gettime();
    HostVectorType xx(x_dp, x_dp+mat.num_rows);
    HostVectorType yy(mat.num_rows, 0.);
    cusp::multiply(mat, xx, yy);
    thrust::copy(yy.begin(), yy.end(), y_dp);
    //std::cout << "ypAx: " << gettime()-T0 << " seconds." << std::endl;
}

template<class LinearOperator,
         class Vector,
         class Preconditioner>
void SystemMatrix::runSolver(LinearOperator& A, Vector& x, Vector& b,
                             Preconditioner& M, escript::SolverBuddy& sb) const
{
    //cusp::verbose_monitor<double> monitor(b, sb.getIterMax(), sb.getTolerance(), sb.getAbsoluteTolerance());
    cusp::default_monitor<double> monitor(b, sb.getIterMax(), sb.getTolerance(), sb.getAbsoluteTolerance());
    int solver = sb.getSolverMethod();
    if (solver == escript::SO_DEFAULT) {
        if (sb.isSymmetric()) {
            solver = escript::SO_METHOD_PCG;
        } else {
            solver = escript::SO_METHOD_BICGSTAB;
        }
    }

    double T0 = gettime();
    switch (solver) {
        case escript::SO_DEFAULT:
        case escript::SO_METHOD_PCG:
            cusp::krylov::cg(A, x, b, monitor, M);
            break;
        case escript::SO_METHOD_CGLS:
            cusp::krylov::cgls(A, x, b, 0., monitor); //shift=0 for now
            break;
        case escript::SO_METHOD_LSQR:
            cusp::krylov::lsqr(A, x, b, cusp::krylov::lsqr_parameters<double>(), monitor);
            break;
        case escript::SO_METHOD_BICGSTAB:
            cusp::krylov::bicgstab(A, x, b, monitor, M);
            break;
        case escript::SO_METHOD_GMRES:
            {
                const int restart = (sb.getRestart()==0 ? 1000 : sb.getRestart());
                if (restart < 1)
                    throw RipleyException("Invalid restart parameter for GMRES");
                cusp::krylov::gmres(A, x, b, restart, monitor, M);
            }
            break;
        case escript::SO_METHOD_PRES20:
            {
                const int restart = 20;
                cusp::krylov::gmres(A, x, b, restart, monitor, M);
            }
            break;
        default:
            throw RipleyException("Unsupported solver.");
    }
    double solvertime = gettime()-T0;

    if (monitor.converged()) {
        if (sb.isVerbose()) {
            std::cout << "Solver converged to " << monitor.relative_tolerance()
                << " relative tolerance after " << monitor.iteration_count()
                << " iterations and " << solvertime << " seconds."<< std::endl;
        }
    } else {
        std::cout << "Solver reached iteration limit "
            << monitor.iteration_limit() << " before converging"
            << " to " << monitor.relative_tolerance() << " rel. tolerance."
            << std::endl;
    }
}

void SystemMatrix::setToSolution(escript::Data& out, escript::Data& in,
                                 bp::object& options) const
{
    if (m_mpiInfo->size > 1) {
        throw RipleyException("solve: ripley's block diagonal matrix "
                              "is incompatible with MPI.");
    }
    if (out.getDataPointSize() != getBlockSize()) {
        throw RipleyException("solve: block size does not match the number of components of solution.");
    } else if (in.getDataPointSize() != getBlockSize()) {
        throw RipleyException("solve: block size does not match the number of components of right hand side.");
    } else if (out.getFunctionSpace() != getColumnFunctionSpace()) {
        throw RipleyException("solve: matrix function space and function space of solution don't match.");
    } else if (in.getFunctionSpace() != getRowFunctionSpace()) {
        throw RipleyException("solve: matrix function space and function space of right hand side don't match.");
    }

    options.attr("resetDiagnostics")();
    escript::SolverBuddy sb = bp::extract<escript::SolverBuddy>(options);
    out.expand();
    in.expand();

    if (sb.isVerbose()) {
        std::cout << "Matrix has " << mat.num_entries << " entries." << std::endl;
    }
    double* out_dp = out.getSampleDataRW(0);
    const double* in_dp = in.getSampleDataRO(0);
    double T0;

    if (sb.getSolverTarget() == escript::SO_TARGET_GPU) {
#ifdef USE_CUDA
        if (cudaDevices.empty()) {
            checkCUDA();
        }

        if (cudaDevices[0] == -1) {
            throw RipleyException("solve: GPU-based solver requested but no "
                                  "CUDA compatible device available.");
        }

        //TODO: give users options...
        if (sb.isVerbose()) {
            std::cout << "Using CUDA device " << cudaDevices[0] << std::endl;
        }
        hipSetDevice(cudaDevices[0]);

        T0 = gettime();
        DeviceVectorType b(in_dp, in_dp+mat.num_rows);
        double host2dev = gettime()-T0;
        if (sb.isVerbose())
            std::cout << "Copy of b: " << host2dev << " seconds." << std::endl;
        if (matrixAltered) {
            T0 = gettime();
            dmat = mat;
            host2dev = gettime()-T0;
            if (sb.isVerbose())
                std::cout << "Copy of A: " << host2dev << " seconds." << std::endl;
            matrixAltered = false;
        }
        DeviceVectorType x(mat.num_rows, 0.);
        if (sb.isVerbose())
            std::cout << "Solving on CUDA device..." << std::endl;

        if (sb.getPreconditioner() == escript::SO_PRECONDITIONER_NONE) {
            cusp::identity_operator<double, cusp::device_memory> M(mat.num_rows, mat.num_rows);
            runSolver(dmat, x, b, M, sb);
        } else if (sb.getPreconditioner() == escript::SO_PRECONDITIONER_JACOBI) {
            if (sb.isVerbose())
                std::cout << "Using Jacobi preconditioner" << std::endl;
            // TODO: This should be cached as well but that's not supported
            // at the moment.
            cusp::precond::diagonal<double, cusp::device_memory> M(dmat);
            runSolver(dmat, x, b, M, sb);
        } else {
            throw RipleyException("Unsupported preconditioner requested.");
        }

        T0 = gettime();
        thrust::copy(x.begin(), x.end(), out_dp);
        const double copyTime = gettime()-T0;
        if (sb.isVerbose())
            std::cout << "Copy of x: " << copyTime << " seconds." << std::endl;
#else
        throw RipleyException("solve: GPU-based solver requested but escript "
                              "not compiled with CUDA.");
#endif
    } else { // CPU
        T0 = gettime();
        HostVectorType b(in_dp, in_dp+mat.num_rows);
        double copytime = gettime()-T0;
        if (sb.isVerbose()) {
            std::cout << "Copy of b: " << copytime << " seconds." << std::endl;
            std::cout << "Solving on the CPU..." << std::endl;
        }
        HostVectorType x(mat.num_rows, 0.);
        if (sb.getPreconditioner() == escript::SO_PRECONDITIONER_NONE) {
            cusp::identity_operator<double, cusp::host_memory> M(mat.num_rows, mat.num_rows);
            runSolver(mat, x, b, M, sb);
        } else if (sb.getPreconditioner() == escript::SO_PRECONDITIONER_JACOBI) {
            if (sb.isVerbose())
                std::cout << "Using Jacobi preconditioner" << std::endl;
            // TODO: This should be cached as well but that's not supported
            // at the moment.
            cusp::precond::diagonal<double, cusp::host_memory> M(mat);
            runSolver(mat, x, b, M, sb);
        } else {
            throw RipleyException("Unsupported preconditioner requested.");
        }

        T0 = gettime();
        thrust::copy(x.begin(), x.end(), out_dp);
        const double copyTime = gettime()-T0;
        if (sb.isVerbose()) {
            std::cout << "Copy of x: " << copyTime << " seconds." << std::endl;
        }
    }
}

void SystemMatrix::nullifyRowsAndCols(escript::Data& row_q,
                                      escript::Data& col_q,
                                      double mdv)
{
    //double T0 = gettime();
    if (col_q.getDataPointSize() != getColumnBlockSize()) {
        throw RipleyException("nullifyRowsAndCols: column block size does not match the number of components of column mask.");
    } else if (row_q.getDataPointSize() != getRowBlockSize()) {
        throw RipleyException("nullifyRowsAndCols: row block size does not match the number of components of row mask.");
    } else if (col_q.getFunctionSpace() != getColumnFunctionSpace()) {
        throw RipleyException("nullifyRowsAndCols: column function space and function space of column mask don't match.");
    } else if (row_q.getFunctionSpace() != getRowFunctionSpace()) {
        throw RipleyException("nullifyRowsAndCols: row function space and function space of row mask don't match.");
    }

    row_q.expand();
    col_q.expand();
    const double* rowMask = row_q.getSampleDataRO(0);
    const double* colMask = col_q.getSampleDataRO(0);
    const int blockSize = getBlockSize();
#pragma omp parallel for
    for (int row=0; row < mat.num_rows; row++) {
        for (int diag=0; diag < mat.diagonal_offsets.size(); diag++) {
            const int col = blockSize*(row/blockSize)+mat.diagonal_offsets[diag]*blockSize;
            if (col >= 0 && col <= mat.num_rows-blockSize) {
                for (int i=0; i<blockSize; i++) {
                    if (rowMask[row] > 0. || colMask[col+i] > 0.) {
                        mat.values(row, diag*blockSize+i) =
                                                        (row==col+i ? mdv : 0);
                    }
                }
            }
        }
    }
    //std::cout << "nullifyRowsAndCols: " << gettime()-T0 << " seconds." << std::endl;
    matrixAltered = true;
}

void SystemMatrix::saveMM(const std::string& filename) const
{
    const int blockSize = getBlockSize();

    std::ofstream f(filename.c_str());
    f << "%%MatrixMarket matrix coordinate real general" << std::endl;
    f << mat.num_rows << " " << mat.num_cols << " " << mat.num_entries << std::endl;
    f.setf(std::ios_base::scientific, std::ios_base::floatfield);
    f.precision(15);
    for (int row=0; row < mat.num_rows; row++) {
        for (int diag=0; diag < mat.diagonal_offsets.size(); diag++) {
            const int col = blockSize*(row/blockSize)+mat.diagonal_offsets[diag]*blockSize;
            if (col >= 0 && col <= mat.num_rows-blockSize) {
                for (int i=0; i<blockSize; i++) {
                    f << row+1 << " " << col+i+1 << " "
                          << mat.values(row, diag*blockSize+i) << std::endl;
                }
            }
        }
    }
}

void SystemMatrix::saveHB(const std::string& filename) const
{
    throw RipleyException("Harwell-Boeing interface not available.");
}

void SystemMatrix::resetValues()
{
    mat.values.values.assign(mat.values.values.size(), 0.);
    matrixAltered = true;
}

}  // end of namespace
